#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

int main(int argc, char * argv[]) {
    int N = atoi(argv[1]);
    int * arr = (int*) calloc(N, sizeof(int));

    size_t size = N * sizeof(int);
    int * dArr;
    hipMalloc(&dArr, size);
    hipMemset(dArr, 0, size);
    hipMemcpy(arr, dArr, size, hipMemcpyDeviceToHost);
    free(arr);
    hipFree(dArr);

    return 0;
}
