#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

int main(int argc, char * argv[]) {
    int N = atoi(argv[1]);

    size_t size = N * sizeof(int);
    int * dArr;
    hipMallocManaged(&dArr, size);
    hipMemset(dArr, 0, size);
    hipDeviceSynchronize();
    hipFree(dArr);

    return 0;
}
